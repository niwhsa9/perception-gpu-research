#include "common.hpp"
#include <iostream>

//Cuda error checking function
bool checkStatus(hipError_t status) {
	if (status != hipSuccess) {
		printf("%s \n", hipGetErrorString(status));
		return true;
	}
    return false;
}

//ceiling division
int ceilDiv(int x, int y) {
    return (x + y - 1) / y;
}