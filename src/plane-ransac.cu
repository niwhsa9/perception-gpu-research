#include "hip/hip_runtime.h"
#include "plane-ransac.hpp"
#include "common.hpp"
#include <stdlib.h>

//These should definitely move to common.hpp
__device__ float getX(GPU_Cloud pc, int index) {
    return pc.data[pc.stride * index + 0];
}

__device__ float getY(GPU_Cloud pc, int index) {
    return pc.data[pc.stride * index + 1];
}

__device__ float getZ(GPU_Cloud pc, int index) {
    return pc.data[pc.stride * index + 2];
}

__device__ float3 getPoint(GPU_Cloud pc, int idx) {
    return float3(getX(pc, idx), getY(pc, idx), getZ(pc, idx));
}

/* 
REQUIRES:
    - GPU data cloud
    - A buffer to write inlier counts for each attempted model
    - A buffer to write the randomly selected points for each attempted model
EFFECTS:

Block: 
    Each block represents an "iteration" of the traditional RANSAC algorithm. 
    That is, every block has a different set of randomly chosen 3 points to 
    evaluate the model with. 
Thread:
    Threads are used to decide how many points are inliers to the model. If
    thread max = 1024 and there are 2048 points, each thread will process 2
    points. Each thread will right the number of inliers to its specific spot
    in shared memory. Threads are synced, and then threads will participate
    in a parallel reduction to give the total number of inliers, which will 
    be returned from the kernel along with the three points that define the
    model tested on this iteration.

*/
__global__ void ransacKernel(GPU_Cloud pc, int* inlierCounts, int* modelPoints) {
    int iteration = blockIdx.x;

    int randIdx0 = modelPoints[iteration*3 + 0];
    int randIdx1 = modelPoints[iteration*3 + 1];
    int randIdx2 = modelPoints[iteration*3 + 2];
    float3 modelPt0 = getPoint(pc, randIdx0);
    float3 modelPt1 = getPoint(pc, randIdx1);
    float3 modelPt2 = getPoint(pc, randIdx2);

    int pointsPerThread = (pc.size + MAX_THREADS - 1) / MAX_THREADS;
    for(int i = 0; i < pointsPerThread; i++) {
        int pointIdx = threadIdx.x * pointsPerThread + i;
        if(pointIdx > pc.size) return; 

        float3 curPt = getPoint(pc, pointIdx);

        //calculate distance of cur pt to the plane formed by the 3 model points

        //add a 0 if inlier, 1 if not 
    }
    
    //parallel reduction

    if(threadIdx.x == 0) {
        inlierCounts[iteration] = __shar
    }
}


RansacPlane::RansacPlane(Vector3d axis, float epsilon, int iterations, float threshold, int pcSize)
: pc(pc), axis(axis), epsilon(epsilon), iterations(iterations), threshold(threshold)  {
    //Set up buffers needed for RANSAC
    hipMalloc(&inlierCounts, sizeof(int) * iterations);
    hipMalloc(&modelPoints, sizeof(int) * iterations * 3);
    
    //Generate random numbers in CPU to use in RANSAC kernel
    int* randomNumsCPU = (int*) alloc(sizeof(int) * iterations* 3);
    for(int i = 0; i < iterations*3; i++) {
        randomNumsCPU[i] = rand() % pcSize;
    }
    hipMemcpy(modelPoints, randomNumsCPU, sizeof(int) * iterations * 3, hipMemcpyHostToDevice);
    free(randomNumsCPU);
}

/*  
EFFECTS:
    1. [GPU] Use the RANSAC kernel to evaluate all the canidate models and report their associated inlier count
    2. [GPU] Select the canidate with the highest score and copy its three model points back to the CPU
    3. [CPU] Use the three model points to produce a plane equation in standard form and return to the user
*/
RansacPlane::Plane RansacPlane::computeModel(GPU_Cloud pc) {
    this->pc = pc;

    int blocks = iterations;
    int threads = MAX_THREADS;
    ransacKernel<<<blocks, threads>>>(pc, inlierCounts, modelPoints);
    checkStatus(hipGetLastError());
    checkStatus(hipDeviceSynchronize());

    return {0, 0, 0, 0};
}

/*
EFFECTS:
    1. Uses the selection computed in computeModel() and the modelPoints of that selection
    to re-calculate the inliers and export them in a list. 
*/
GPU_Indicies RansacPlane::getInliers() {
    return {nullptr, 0};
}


RansacPlane::~RansacPlane() {
    hipFree(inlierCounts);
    hipFree(modelPoints);
}